#include "hip/hip_runtime.h"
#include "field.h"

extern "C" __global__ void upsweep_kernel(unsigned int *src, unsigned int *dst, int n)
{
    int idx0 = threadIdx.x + (blockIdx.x * blockDim.x);
    if (idx0 >= n)
        return;
    int idx1 = idx0 + n;
    // a0' = a0 * a1 .
    src[idx1] = dst[idx0];
    dst[idx0] = mul31(dst[idx0], dst[idx1]);
}

extern "C" __global__ void downsweep_kernel(unsigned int *src, unsigned int *dst, int n)
{
    int idx0 = threadIdx.x + (blockIdx.x * blockDim.x);
    if (idx0 >= n)
        return;
    int idx1 = idx0 + n;
    // a0'' = inv(a0') * a1 .
    // a1'' = inv(a0') * a0 .
    unsigned int inva0a1 = dst[idx0];
    dst[idx0] = mul31(inva0a1, dst[idx1]);
    dst[idx1] = mul31(inva0a1, src[idx1]);
}